#include "hip/hip_runtime.h"
#include<cstdio>
#include<omp.h>
#include"globals.h"
#include"cudaGlobals.h"
#include"hipError_t.h"
#define sq(x) ((x)*(x))
//tutaj tez zrobic te optymalizacje zeby 2x nie czytac najlepiej wzgledem wymiaru Xowego
#define THREADS 128


texture<float> texPii;
texture<float> texTau;
texture<float> texaReal;
texture<float> texbReal;
texture<float> texaImag;
texture<float> texbImag;

#if __CUDA_ARCH__ >= 300
__global__ void kernelGenerate(int const rSize, int const nPiiTau, real const * const pii, real const * const tau, 
		real const * const aReal, real const * const aImag, real const * const bReal, real const * const bImag,
		int const * const NmaxTable, real * const II) {
	volatile __shared__ real realis[4];
	volatile __shared__ real imaginalis[4];
	const int Nmax = NmaxTable[blockIdx.y];
	int index  = blockIdx.x * nPiiTau + threadIdx.x;
	int indexY = blockIdx.y * nPiiTau + threadIdx.x;
	real r=0.0f;
	real i=0.0f;
	if(threadIdx.x < Nmax) {
		//const real pi = pii[index];
		//const real ta = tau[index];
		const real pi = tex1Dfetch(texPii, index);
		const real ta = tex1Dfetch(texTau, index);
		//r = aReal[indexY] * pi + bReal[indexY] * ta;
		//i = aImag[indexY] * pi + bImag[indexY] * ta;
		r = tex1Dfetch(texaReal,indexY) * pi + tex1Dfetch(texbReal,indexY) * ta;
		i = tex1Dfetch(texaImag,indexY) * pi + tex1Dfetch(texbImag,indexY) * ta;
		for(int id = THREADS ; id < Nmax ; id+=THREADS) {
			if(threadIdx.x + id < Nmax) {
				index += THREADS; 
				indexY += THREADS;
				//const real pi = pii[index];
				//const real ta = tau[index];
				const real pi = tex1Dfetch(texPii, index);
				const real ta = tex1Dfetch(texTau, index);
				//r += aReal[indexY] * pi + bReal[indexY] * ta;
				//i += aImag[indexY] * pi + bImag[indexY] * ta;
				r += tex1Dfetch(texaReal,indexY) * pi + tex1Dfetch(texbReal,indexY) * ta;
				i += tex1Dfetch(texaImag,indexY) * pi + tex1Dfetch(texbImag,indexY) * ta;
			}
		}
	}

	//butterfly reduction across warp
	for (int j=16; j>=1; j/=2) {
		r += __shfl_xor(r , j, 32);
		i += __shfl_xor(i , j, 32);
	}
	//further reduction across block
	if(threadIdx.x % 32 == 0) {
		realis[threadIdx.x>>5] = r;
		imaginalis[threadIdx.x>>5] = i;
	}
	__syncthreads();
	if(threadIdx.x <2) {
		realis[threadIdx.x] += realis[threadIdx.x+2];
		imaginalis[threadIdx.x] += imaginalis[threadIdx.x+2];
		realis[threadIdx.x] += realis[threadIdx.x+1];
		imaginalis[threadIdx.x] += imaginalis[threadIdx.x+1];
	}

	if(threadIdx.x==0)
		II[blockIdx.x + blockIdx.y*gridDim.x] = sq(realis[0]) + sq(imaginalis[0]);
}
#endif
#if __CUDA_ARCH__ < 300
__global__ void kernelGenerate(int const rSize, int const nPiiTau, real const * const pii, real const * const tau, real const * const aReal, real const * const aImag, real const * const bReal, real const * const bImag, int const * const NmaxTable, real * const II) {
	volatile __shared__ real realis[THREADS];
	volatile __shared__ real imaginalis[THREADS];
	const int Nmax = NmaxTable[blockIdx.y];
	int index = blockIdx.x * nPiiTau + threadIdx.x;
	int indexY = blockIdx.y * nPiiTau + threadIdx.x; 
	real r=0.0f;
	real i=0.0f;
	if(threadIdx.x < Nmax) {
		const real pi = pii[index];
		const real ta = tau[index];
		//const real pi = tex1Dfetch(texPii, index);
		//const real ta = tex1Dfetch(texTau, index);
		r = aReal[indexY] * pi + bReal[indexY] * ta;
		i = aImag[indexY] * pi + bImag[indexY] * ta;
		//r = tex1Dfetch(texaReal,indexY) * pi + tex1Dfetch(texbReal,indexY) * ta;
		//i = tex1Dfetch(texaImag,indexY) * pi + tex1Dfetch(texbImag,indexY) * ta;
		for(int id = THREADS ; id < Nmax ; id+=THREADS) {
			if(threadIdx.x + id < Nmax) {
				index += THREADS; 
				indexY += THREADS;
				const real pi = pii[index];
				const real ta = tau[index];
				//const real pi = tex1Dfetch(texPii, index);
				//const real ta = tex1Dfetch(texTau, index);
				r += aReal[indexY] * pi + bReal[indexY] * ta;
				i += aImag[indexY] * pi + bImag[indexY] * ta;
				//r += tex1Dfetch(texaReal,indexY) * pi + tex1Dfetch(texbReal,indexY) * ta;
				//i += tex1Dfetch(texaImag,indexY) * pi + tex1Dfetch(texbImag,indexY) * ta;
			}
		}
	}
	if ( threadIdx.x < Nmax ) {
		realis[threadIdx.x] = r;
		imaginalis[threadIdx.x] = i;
	}
	else {
		realis[threadIdx.x]=0.0f;
		imaginalis[threadIdx.x]=0.0f;
	}
	__syncthreads();
	if(threadIdx.x < 64 ) {
		realis[threadIdx.x]+=realis[threadIdx.x+64];
		imaginalis[threadIdx.x]+=imaginalis[threadIdx.x+64];
	}
	__syncthreads();
	if(threadIdx.x < 32) {
		realis[threadIdx.x]+=realis[threadIdx.x+32];
		realis[threadIdx.x]+=realis[threadIdx.x+16];
		realis[threadIdx.x]+=realis[threadIdx.x+8];
		realis[threadIdx.x]+=realis[threadIdx.x+4];
		realis[threadIdx.x]+=realis[threadIdx.x+2];
		realis[threadIdx.x]+=realis[threadIdx.x+1];
		imaginalis[threadIdx.x]+=imaginalis[threadIdx.x+32];
		imaginalis[threadIdx.x]+=imaginalis[threadIdx.x+16];
		imaginalis[threadIdx.x]+=imaginalis[threadIdx.x+8];
		imaginalis[threadIdx.x]+=imaginalis[threadIdx.x+4];
		imaginalis[threadIdx.x]+=imaginalis[threadIdx.x+2];
		imaginalis[threadIdx.x]+=imaginalis[threadIdx.x+1];
		if(threadIdx.x==0)
			II[blockIdx.x + blockIdx.y*gridDim.x] = sq(realis[0]) + sq(imaginalis[0]);
	}
}


#endif

void cudaGenerate(int rSize, int pattern_length, int * Nmax, real * pii, int nPiiTau,  real * tau, real * aReal, real * aImag, real * bReal, real * bImag, real * II, int polarization ) {
	CudaSafeCall(hipStreamCreate(&stream[polarization]));
	CudaSafeCall(hipMalloc((void**)&devPii[polarization]  , nPiiTau*pattern_length*sizeof(real)));
	CudaSafeCall(hipMalloc((void**)&devTau[polarization]  , nPiiTau*pattern_length*sizeof(real)));
	CudaSafeCall(hipMalloc((void**)&devAReal[polarization], rSize*nPiiTau*sizeof(real)));
	CudaSafeCall(hipMalloc((void**)&devBReal[polarization], rSize*nPiiTau*sizeof(real)));
	CudaSafeCall(hipMalloc((void**)&devAImag[polarization], rSize*nPiiTau*sizeof(real)));
	CudaSafeCall(hipMalloc((void**)&devBImag[polarization], rSize*nPiiTau*sizeof(real)));
	CudaSafeCall(hipMalloc((void**)&devII[polarization]   , rSize*pattern_length*sizeof(real)));
	CudaSafeCall(hipMalloc((void**)&devNmax[polarization] , rSize*sizeof(int)));

	CudaSafeCall(hipMemcpyAsync(devPii[polarization]  , pii  , nPiiTau*pattern_length*sizeof(real) , hipMemcpyHostToDevice , stream[polarization]));
	CudaSafeCall(hipMemcpyAsync(devTau[polarization]  , tau  , nPiiTau*pattern_length*sizeof(real) , hipMemcpyHostToDevice , stream[polarization]));
	CudaSafeCall(hipMemcpyAsync(devNmax[polarization] , Nmax , rSize*sizeof(int)                   , hipMemcpyHostToDevice , stream[polarization]));
	CudaSafeCall( hipBindTexture( NULL, texPii, devPii[polarization], nPiiTau*pattern_length*sizeof(real)));
	CudaSafeCall( hipBindTexture( NULL, texTau, devTau[polarization], nPiiTau*pattern_length*sizeof(real)));

	CudaSafeCall(hipMemcpyAsync(devAReal[polarization], aReal, rSize*nPiiTau*sizeof(real), hipMemcpyHostToDevice, stream[polarization]));
	CudaSafeCall(hipMemcpyAsync(devBReal[polarization], bReal, rSize*nPiiTau*sizeof(real), hipMemcpyHostToDevice, stream[polarization]));
	CudaSafeCall(hipMemcpyAsync(devAImag[polarization], aImag, rSize*nPiiTau*sizeof(real), hipMemcpyHostToDevice, stream[polarization]));
	CudaSafeCall(hipMemcpyAsync(devBImag[polarization], bImag, rSize*nPiiTau*sizeof(real), hipMemcpyHostToDevice, stream[polarization]));
	CudaSafeCall( hipBindTexture( NULL, texaReal, devAReal[polarization], rSize*nPiiTau*sizeof(real)));
	CudaSafeCall( hipBindTexture( NULL, texbReal, devBReal[polarization], rSize*nPiiTau*sizeof(real)));
	CudaSafeCall( hipBindTexture( NULL, texaImag, devAImag[polarization], rSize*nPiiTau*sizeof(real)));
	CudaSafeCall( hipBindTexture( NULL, texbImag, devBImag[polarization], rSize*nPiiTau*sizeof(real)));

#ifdef GF580
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernelGenerate), hipFuncCachePreferL1);
#endif //GF580
#ifdef GF680
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernelGenerate), hipFuncCachePreferShared);
#endif //GF680
	kernelGenerate<<<dim3(pattern_length,rSize,1), THREADS, 0, stream[polarization]>>>(rSize, nPiiTau, devPii[polarization], devTau[polarization],
			devAReal[polarization], devAImag[polarization], devBReal[polarization], devBImag[polarization],
			devNmax[polarization], devII[polarization]);

	CudaSafeCall(hipMemcpyAsync(II, devII[polarization], rSize*pattern_length*sizeof(real), hipMemcpyDeviceToHost, stream[polarization]));
	CudaSafeCall(hipUnbindTexture( texPii));
	CudaSafeCall(hipUnbindTexture( texTau));
	CudaSafeCall(hipUnbindTexture( texaReal));
	CudaSafeCall(hipUnbindTexture( texbReal));
	CudaSafeCall(hipUnbindTexture( texbImag));
	CudaSafeCall(hipUnbindTexture( texaImag));
}
#undef sq



