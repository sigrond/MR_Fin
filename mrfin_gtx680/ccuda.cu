#include"globals.h"
#include"cudaGlobals.h"
#include"hipError_t.h"
real * devPii[2];
real * devTau[2];
real * devAReal[2];
real * devAImag[2];
real * devBReal[2];
real * devBImag[2];
real * devII[2];
int * devNmax[2];
hipStream_t stream[2];
real * devReferences[2];
real * devErr[2];
real * devPatterns[2];
real * devInvRSquare[2];
real * devPSquare[2];
hipStream_t streamRef[2];
real * devMin[2];
real * devMax[2];
int * devMinIndex[2];
real * devMedian[2];
int * devOut;


void freeCudaPointer(void ** pointer) {
	CudaSafeCall(hipHostFree(*pointer));
}
void allocCudaPointer(void ** pointer, size_t size) {
	CudaSafeCall(hipHostMalloc((void**)pointer, size));
}
	
void mallocCudaReferences(int i, int const mPatterns, int const nPatterns, int const mReferences, int const nReferences ) {
			CudaSafeCall(hipStreamCreate(&streamRef[i]));
			CudaSafeCall(hipMalloc((void**)&devPatterns[i], mPatterns*nPatterns*sizeof(real)));
			CudaSafeCall(hipMalloc((void**)&devReferences[i],mReferences*nReferences*sizeof(real)));
			CudaSafeCall(hipMalloc((void**)&devInvRSquare[i], mReferences*sizeof(real)));
			CudaSafeCall(hipMalloc((void**)&devPSquare[i], mPatterns*sizeof(real)));
			CudaSafeCall(hipMalloc((void**)&devErr[i], mPatterns*mReferences*sizeof(real)));
			CudaSafeCall(hipMalloc((void**)&devMin[i], mPatterns*sizeof(real)));
			CudaSafeCall(hipMalloc((void**)&devMax[i], mPatterns*sizeof(real))); //TODO: czy rozmiar dobry? (04.04.13 by szmigacz)
			CudaSafeCall(hipMalloc((void**)&devMinIndex[i], mPatterns*sizeof(int)));
			CudaSafeCall(hipMalloc((void**)&devMedian[i], mPatterns*sizeof(real)));

}

void freeCudaMemory() {
	#ifdef CUDA
		for(int i=0;i<2;i++) {
			CudaSafeCall(hipStreamSynchronize(stream[i]));
		}
	
		for(int i=0;i<2;i++) {
			CudaSafeCall(hipFree(devPii[i]));
			CudaSafeCall(hipFree(devTau[i]));
			CudaSafeCall(hipFree(devAReal[i]));
			CudaSafeCall(hipFree(devAImag[i]));
			CudaSafeCall(hipFree(devBReal[i]));
			CudaSafeCall(hipFree(devBImag[i]));
			CudaSafeCall(hipFree(devII[i]));
			CudaSafeCall(hipFree(devNmax[i]));
			CudaSafeCall(hipStreamDestroy(stream[i]));
		}
	#endif //CUDA
}

void freeCudaRefMemory() {
	#ifdef CUDA
		for(int i=0;i<2;i++) {
			CudaSafeCall(hipStreamSynchronize(streamRef[i]));
		}
		for(int i=0;i<2;i++) {
	
			CudaSafeCall(hipFree(devInvRSquare[i]));
			CudaSafeCall(hipFree(devPSquare[i]));
			CudaSafeCall(hipFree(devErr[i]));
			CudaSafeCall(hipFree(devPatterns[i]));
			CudaSafeCall(hipFree(devReferences[i]));
			CudaSafeCall(hipFree(devMin[i]));
			CudaSafeCall(hipFree(devMax[i]));
			CudaSafeCall(hipFree(devMinIndex[i]));
			CudaSafeCall(hipFree(devMedian[i]));
			CudaSafeCall(hipStreamDestroy(streamRef[i]));
		}
	#endif //CUDA
}

void cudaFinalize() {
	#ifdef CUDA
		CudaSafeCall(hipDeviceSynchronize());
	#endif //CUDA
}
void cuda1stPolarizationSync() {
	#ifdef CUDA
		CudaSafeCall(hipStreamSynchronize(streamRef[0]));
		CudaSafeCall(hipStreamSynchronize(streamRef[1]));
	#endif //CUDA
}

void freeCudaMemoryMin() {
	#ifdef CUDA
			CudaSafeCall(hipFree(devOut));
	#endif //CUDA
}
